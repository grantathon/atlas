#include "householder.cuh"
#include "hipblas.h"
#include "aux.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define IDX2C(i,j,ld) (((j)*(ld))+(i))

using namespace std;

// Reduce the matrix to a tridiagonal matrix via Householder transformations
int BlockPairReduction(float *q, float *column, float *block_pair, int dim)
{
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    float *dBlockPair;
    
    hipMalloc(&dBlockPair, (size_t)dim*dim*sizeof(*block_pair)); CUDA_CHECK;
    
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }
    
    stat = hipblasSetMatrix (dim, dim, sizeof(*block_pair), block_pair, dim, dBlockPair, dim);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download failed");
        hipFree (dBlockPair);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    
    // TODO: Call cuBLAS functions to solve for first column
    //      of reduced matrix (b) and the next Q (q).
    
    //modify (handle, dBlockPair, M, N, 1, 2, 16.0f, 12.0f);
    
    stat = hipblasGetMatrix (dim, dim, sizeof(*block_pair), dBlockPair, dim, block_pair, dim);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data upload failed");
        hipFree (dBlockPair);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    
    hipFree (dBlockPair);
    hipblasDestroy(handle);
    
    // TODO: The function should return the Q and the first column.
    
    return 0;
}

